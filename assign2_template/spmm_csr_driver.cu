#include "hip/hip_runtime.h"
/*
 * spmm_csr_driver.cu
 * Copyright (C) 2018
 *  P Sadayappan (saday) <psaday@gmail.com>
 *  Aravind SUKUMARAN RAJAM (asr) <aravind_sr@outlook.com>
 *
 * Distributed under terms of the GNU LGPL3 license.
 */

#include "mm_helper.hpp"
#include "sparse_representation.hpp"
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define TILE_WIDTH 32

void check_dmat(double* a, double *b,  int n,  int K, bool quit_on_err = true ) {
    for ( int i = 0; i < n; ++i) {
        for ( int k = 0; k < K; ++k) {
            if(std::abs(a[i * K + k] - b[i * K + k]) > 1e-1) {
                std::cerr << "Possible error at " << i << std::endl;

                if(quit_on_err) {
                    exit(-1);
                }
            }
        }
    }

    if(quit_on_err)
        std::cout << "Verification succeeded\n";
    else
        std::cout << "Check error messages to see if verification succeeded. (No error msg == success)\n";
}

static  int g_seed = 0X4B1D;
inline int fastrand() {
    g_seed = (214013 * g_seed + 2531011);
    return (g_seed >> 16) & 0x7FFF;
}

void init_dmat(double *a,  int n,  int K, double offset) {
    for ( int i = 0; i < n; ++i) {
        for ( int k = 0; k < K; ++k) {
            a[i * K + k]  = i * K + k + offset;
            //a[i * K + j]  = fastrand() + offset;
        }
    }
}

void print_dmat(double *a,  int n,  int K) {
    for ( int i = 0; i < n; ++i) {
        for ( int j = 0; j < K; ++j) {
            std::cout << a[i * K + j]   << ' ';
        }
        std::cout << '\n';
    }
}

void print_CSR(CSR &mat) {
    for ( int r = 0; r < mat.nrows; ++r) {
         int row_start = mat.row_indx[r];
         int row_end = mat.row_indx[r + 1];
        for ( int j = row_start; j < row_end; ++j) {
             int col_id = mat.col_id[j];
            double val = mat.values[j];

	    std::cout << r << ' ' << col_id << ' ' <<  val << '\n';
        }
    }
}

void host_csr_spmm(CSR &mat, double * dmat_in, double * dmat_out,  int K) {
    for ( int r = 0; r < mat.nrows; ++r) {
         int row_start = mat.row_indx[r];
         int row_end = mat.row_indx[r + 1];

        for ( int k = 0; k < K; ++k) {
            dmat_out[r * K + k] = 0;
        }

        for ( int j = row_start; j < row_end; ++j) {
             int col_id = mat.col_id[j];
            double val = mat.values[j];

            for ( int k = 0; k < K; ++k) {
                dmat_out[r * K + k] += val * dmat_in[col_id * K + k];
            }
        }

    }
}

//Emin Code start
__global__ void dev_csr_spmm( int * deviceCSRrow_indx ,  int * deviceCSRcol_id  ,  int * deviceCSRvalues,
   double * dmat_in_device, double* dmat_out_device ,  int K ,  int device_nrows ){


      int iy= blockIdx.y*blockDim.y + threadIdx.y ;
      int ix= blockIdx.x*blockDim.x+  threadIdx.x ;

      //int numberOfRowCSR = A.nrows;
      int numberOfRowCSR = device_nrows ;
      int colId;
      //const int row = blockIdx.x * blockDim.x + threadIdx.x ;

      if ( iy < numberOfRowCSR && ix < K) {

        double sum=0.0;

        // int row_start = A.row_indx[iy] ;
         int row_start = deviceCSRrow_indx[iy];
        // int row_end = A.row_indx[iy + 1] ;
         int row_end = deviceCSRrow_indx[iy+1] ;


        for (int i = row_start; i < row_end; i++) {
          /* code */
          //colId= A.col_id[i] ;
          colId = deviceCSRcol_id[i] ;
          //sum += A.values[i] * dmat_in_device[colId * K + ix] ;
          sum += deviceCSRvalues[i] ;
        }

        //dmat_out[ix][iy] = sum ;
        dmat_out_device[ix * K + iy] = sum ;
      }

}




int main(int argc, char *argv[]) {
    if(argc < 3) {
        std::cerr << "usage ./exec inputfile K  " << std::endl;
        exit(-1);
    }

     int K = std::atoi(argv[2]);
    CSR mat = read_matrix_market_to_CSR(argv[1]);
    //print_CSR(mat);
    std::cout << mat.nrows << ' ' << mat.ncols << ' ' << mat.nnz << ' ' << K << '\n';

    double *dmat_in = (double*)malloc(mat.ncols * K  * sizeof(double));
    double *dmat_out = (double*)malloc(mat.nrows * K * sizeof(double));
    double *dmat_out_GPU = (double*)malloc(mat.nrows * K * sizeof(double));

    init_dmat(dmat_in, mat.ncols, K,  1.0);
    //print_dmat(dmat_in, mat.ncols, K);

    host_csr_spmm(mat, dmat_in, dmat_out, K);


    //Prepeare for Kernel
    //CSR *temMat;
    //temMat->nrows = mat.nrows ;
    //temMat.->ncols = mat.ncols ;
    //temMat.->nnz = mat.nnz ;

     int* deviceCSRrow_indx;
     int* deviceCSRcol_id;
    double* deviceCSRvalues;

     int* device_nrows;
     int* device_ncols;
     int* device_nnz;

     int a=1;

    hipMalloc((void**) &deviceCSRrow_indx ,(mat.nrows +1) * sizeof(int)) ;
    hipMalloc((void**) &deviceCSRcol_id , mat.ncols * sizeof(int)) ;
    hipMalloc((void**) &deviceCSRvalues , mat.nnz * sizeof(double)) ;

    hipMalloc((void**) &device_nrows, mat.nrows , a*sizeof(int));
    hipMalloc((void**) &device_ncols, mat.ncols , a*sizeof(int));
    hipMalloc((void**) &device_nnz, mat.nnz , a*sizeof(int));

    //hipMalloc((void**) &(temMat->values) , mat.nnz * sizeof(double)) ;
    //hipMalloc((void**) &(temMat->row_indx) , mat.nrows * sizeof( int)) ;
    //hipMalloc((void**) &(temMat->col_id) , mat.ncols * sizeof( int)) ;

    //hipMalloc((void**) &(temMat->nrows) , sizeof( int)) ;
    //hipMalloc((void**) &(temMat->ncols) , sizeof( int)) ;
    //hipMalloc((void**) &(temMat->nnz) , sizeof( int)) ;

    //Initialize device addresses since it can not be accessed directly
    //hipMemcpy(temMat->values , mat.values , mat.nnz * sizeof(double) , hipMemcpyHostToDevice) ;
    //hipMemcpy(temMat->row_indx , mat.row_indx , mat.nrows * sizeof( int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(temMat->col_id , mat.col_id , mat.ncols * sizeof( int) , hipMemcpyHostToDevice) ;

    hipMemcpy(deviceCSRrow_indx , mat.row_indx ,  mat.nrows * sizeof( int) , hipMemcpyHostToDevice) ;
    hipMemcpy(deviceCSRcol_id, mat.col_id , mat.ncols * sizeof( int) , hipMemcpyHostToDevice) ;
    hipMemcpy(deviceCSRvalues , mat.values , mat.nnz * sizeof(double) , hipMemcpyHostToDevice) ;

    hipMemcpy(device_nrows , mat.nrows , sizeof( int) , hipMemcpyHostToDevice) ;
    hipMemcpy(device_ncols , mat.ncols , sizeof( int) , hipMemcpyHostToDevice) ;
    hipMemcpy(device_nnz   , mat.nnz   , sizeof( int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(temMat->nrows , mat.nrows , 1*sizeof( int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(temMat->ncols , mat.ncols , 1*sizeof( int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(temMat->nnz , mat.nnz , 1*sizeof( int) , hipMemcpyHostToDevice) ;

    //CSR A;
    //hipMemcpyToSymbol(HIP_SYMBOL( A ), temMat , sizeof(CSR)) ;

    double *dmat_in_device ;
    hipMalloc((void**) &dmat_in_device , mat.ncols * K * sizeof(double)) ;

    double *dmat_out_device ;
    hipMalloc((void**) &dmat_out_device, mat.nrows * K * sizeof(double)) ;

    //copy to device
    hipMemcpy( dmat_in_device , dmat_in , mat.ncols * K * sizeof(double) , hipMemcpyHostToDevice ) ;
    hipMemcpy( dmat_out_device, dmat_out, mat.nrows * K * sizeof(double) , hipMemcpyHostToDevice ) ;


    //Initialize the Grid and Block Dimension

    dim3 dimGrid((K-1) / TILE_WIDTH + 1 , (mat.nrows -1)/ TILE_WIDTH +1 , 1  ) ;
    dim3 dimBlock(TILE_WIDTH , TILE_WIDTH , 1) ;

    dev_csr_spmm<<<dimGrid , dimBlock>>>(deviceCSRrow_indx, deviceCSRcol_id, deviceCSRvalues , dmat_in_device , dmat_out_device , K , device_nrows) ;

    hipMemcpy(dmat_out_GPU , dmat_out_device ,mat.nrows * K * sizeof(double) , hipMemcpyDeviceToHost ) ;


    //td::cout << "replace one argument to the below function with the values from gpu " << std::endl;
    check_dmat(dmat_out, dmat_out_GPU, mat.nrows, K);

    //print_dmat(dmat_out, mat.nrows, K);


    free(mat.row_indx);
    free(mat.col_id);
    free(mat.values);

    hipFree(deviceCSRrow_indx) ;
    hipFree(deviceCSRcol_id) ;
    hipFree(deviceCSRvalues) ;

    hipFree(device_nrows) ;
    hipFree(device_ncols) ;
    hipFree(device_nnz ) ;
    return 0;
}
