#include "hip/hip_runtime.h"
/*
 * spmm_csr_driver.cu
 * Copyright (C) 2018
 *  P Sadayappan (saday) <psaday@gmail.com>
 *  Aravind SUKUMARAN RAJAM (asr) <aravind_sr@outlook.com>
 *
 * Distributed under terms of the GNU LGPL3 license.
 */

#include "mm_helper.hpp"
#include "sparse_representation.hpp"
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

//#define TILE_WIDTH 32

void check_dmat(double* a, double *b,  int n,  int K, bool quit_on_err = true ) {
    for ( int i = 0; i < n; ++i) {
        for ( int k = 0; k < K; ++k) {
            if(std::abs(a[i * K + k] - b[i * K + k]) > 1e-1) {
                std::cerr << "Possible error at " << i << std::endl;

                if(quit_on_err) {
                    exit(-1);
                }
            }
        }
    }

    if(quit_on_err)
        std::cout << "Verification succeeded\n";
    else
        std::cout << "Check error messages to see if verification succeeded. (No error msg == success)\n";
}

static  int g_seed = 0X4B1D;
inline int fastrand() {
    g_seed = (214013 * g_seed + 2531011);
    return (g_seed >> 16) & 0x7FFF;
}

void init_dmat(double *a,  int n,  int K, double offset) {
    for ( int i = 0; i < n; ++i) {
        for ( int k = 0; k < K; ++k) {
            a[i * K + k]  = i * K + k + offset;
            //a[i * K + j]  = fastrand() + offset;
        }
    }
}

void print_dmat(double *a,  int n,  int K) {
    for ( int i = 0; i < n; ++i) {
        for ( int j = 0; j < K; ++j) {
            std::cout << a[i * K + j]   << ' ';
        }
        std::cout << '\n';
    }
}

void print_CSR(CSR &mat) {
    for ( int r = 0; r < mat.nrows; ++r) {
         int row_start = mat.row_indx[r];
         int row_end = mat.row_indx[r + 1];
        for ( int j = row_start; j < row_end; ++j) {
             int col_id = mat.col_id[j];
            double val = mat.values[j];

	    std::cout << r << ' ' << col_id << ' ' <<  val << '\n';
        }
    }
}

void host_csr_spmm(CSR &mat, double * dmat_in, double * dmat_out,  int K) {
    for ( int r = 0; r < mat.nrows; ++r) {
         int row_start = mat.row_indx[r];
         int row_end = mat.row_indx[r + 1];

        for ( int k = 0; k < K; ++k) {
            dmat_out[r * K + k] = 0;
        }

        for ( int j = row_start; j < row_end; ++j) {
            int col_id = mat.col_id[j];
            double val = mat.values[j];

            for ( int k = 0; k < K; ++k) {
                dmat_out[r * K + k] += val * dmat_in[col_id * K + k];
            }
        }

    }
}

//Emin Code start
__global__ void dev_csr_spmm(unsigned int * deviceCSRrow_indx , unsigned int * deviceCSRcol_id  ,  double * deviceCSRvalues,
   double * dmat_in_device, double* dmat_out_device ,  int K , unsigned int device_nrows ){


      //int row= blockIdx.y*blockDim.y + threadIdx.y ;
      const int row=blockIdx.y;
      const int col= blockIdx.x * blockDim.x + threadIdx.x ;


      unsigned int numberOfRowCSR = device_nrows ;

      //const int row = blockIdx.x * blockDim.x + threadIdx.x ;
      //printf(" Rows = %d thread %d , block %d \n", numberOfRowCSR,  col , row);

      if ( (row < numberOfRowCSR) && (col < K) ) {

            //printf(" thread %d , block %d \n",  col , row);

            double sum=0;
            int colId;

            // int row_start = A.row_indx[iy] ;
             unsigned int row_start = deviceCSRrow_indx[row];
             //printf(" row_start = %d thread %d , block %d \n", row_start,  col , row);
            // int row_end = A.row_indx[iy + 1] ;
             unsigned int row_end = deviceCSRrow_indx[row+1] ;
             //printf(" row_end = %d thread %d , block %d \n", row_end,  col , row);

             dmat_out_device[row * K + col] =0;

            for ( int element = row_start; element < row_end; element++) {
                  /* code */

                  //colId= A.col_id[i] ;
                  colId = deviceCSRcol_id[element] ;
                  //printf(" colId = %d thread %d , block %d \n", colId,  col , row);

                  double value = deviceCSRvalues[element] ;
                  double value2 = dmat_in_device[colId * K + col] ;

                  //printf(" value %d  thread %d , block %d \n", value,  col , row);

                  sum = sum +  value * value2 ;

                  //printf(" sum =  %d ,thread %d , block %d", sum, col , row);
            }
            //__synctreads();
            //dmat_out[ix][iy] = sum ;
            //printf(" sum = %d thread %d , block %d \n", sum,  col , row);
            dmat_out_device[row * K + col] = sum ;
            //printf("dvice matrix %d\n", dmat_out_device[row * K + col] );
      }

}




int main(int argc, char *argv[]) {
    if(argc < 3) {
        std::cerr << "usage ./exec inputfile K  " << std::endl;
        exit(-1);
    }

    int K = std::atoi(argv[2]);
    CSR mat = read_matrix_market_to_CSR(argv[1]);
    //print_CSR(mat);

    int TILE_WIDTH = K +1 ;

    //Cuda Events
    // events for timing
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent) ;
    hipEventCreate(&stopEvent)  ;


    hipEvent_t startEventMemKer , stopEventMemKer ;
    hipEventCreate(&startEventMemKer);
    hipEventCreate(&stopEventMemKer) ;




    //Lets implement pinned memory
    CSR pinnedMat;
    hipHostAlloc(&pinnedMat.row_indx , (mat.nrows +1)* sizeof(unsigned int), hipHostMallocMapped ) ;
    hipHostAlloc(&pinnedMat.col_id , mat.nnz * sizeof(unsigned int) , hipHostMallocMapped) ;
    hipHostAlloc(&pinnedMat.values , mat.nnz * sizeof(double), hipHostMallocMapped) ;

    memcpy(pinnedMat.row_indx , mat.row_indx ,(mat.nrows +1)* sizeof(unsigned int)) ;
    memcpy(pinnedMat.col_id , mat.col_id ,mat.nnz * sizeof(unsigned int) ) ;
    memcpy(pinnedMat.values , mat.values ,mat.nnz * sizeof(double)) ;

    pinnedMat.nrows=mat.nrows ;
    pinnedMat.ncols=mat.ncols ;
    pinnedMat.nnz = mat.nnz ;

    std::cout << mat.nrows << ' ' << mat.ncols << ' ' << mat.nnz << ' ' << K << '\n';

    double *dmat_in = (double*)malloc(mat.ncols * K  * sizeof(double));
    double *dmat_out = (double*)malloc(mat.nrows * K * sizeof(double));
    double *dmat_out_GPU = (double*)malloc(mat.nrows * K * sizeof(double));

    init_dmat(dmat_in, mat.ncols, K,  1.0);
    //print_dmat(dmat_in, mat.ncols, K);

    host_csr_spmm(mat, dmat_in, dmat_out, K);




    unsigned int* deviceCSRrow_indx;
    unsigned int* deviceCSRcol_id;
    double* deviceCSRvalues;


    hipMalloc((void**) &deviceCSRrow_indx ,(mat.nrows +1) * sizeof(unsigned int)) ;
    hipMalloc((void**) &deviceCSRcol_id , mat.nnz * sizeof(unsigned int)) ;
    hipMalloc((void**) &deviceCSRvalues , mat.nnz * sizeof(double)) ;

    double *dmat_in_device ;
    hipMalloc((void**) &dmat_in_device , mat.ncols * K * sizeof(double)) ;

    double *dmat_out_device ;
    hipMalloc((void**) &dmat_out_device, mat.nrows * K * sizeof(double)) ;

    //We want to use pinned memory

    //hipMemcpy(deviceCSRrow_indx , mat.row_indx ,  (mat.nrows+1) * sizeof(unsigned int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(deviceCSRcol_id, mat.col_id , mat.nnz * sizeof(unsigned int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(deviceCSRvalues , mat.values , mat.nnz * sizeof(double) , hipMemcpyHostToDevice) ;

    hipEventRecord(startEventMemKer, 0);


    //hipStream_t stream;

    //hipMemcpy(deviceCSRrow_indx , pinnedMat.row_indx ,(mat.nrows+1) * sizeof(unsigned int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(deviceCSRcol_id , pinnedMat.col_id , mat.nnz * sizeof(unsigned int) , hipMemcpyHostToDevice ) ;
    //hipMemcpy(deviceCSRvalues , pinnedMat.values , mat.nnz * sizeof(double) , hipMemcpyHostToDevice)  ;

    //copy to device
    //hipMemcpy( dmat_in_device , dmat_in , mat.ncols * K * sizeof(double) , hipMemcpyHostToDevice ) ;
    //hipMemcpy( dmat_out_device, dmat_out, mat.nrows * K * sizeof(double) , hipMemcpyHostToDevice ) ;

    hipMemcpy(deviceCSRrow_indx , pinnedMat.row_indx ,(mat.nrows+1) * sizeof(unsigned int) , hipMemcpyHostToDevice );
    hipMemcpy(deviceCSRcol_id , pinnedMat.col_id , mat.nnz * sizeof(unsigned int) , hipMemcpyHostToDevice );
    hipMemcpy(deviceCSRvalues , pinnedMat.values , mat.nnz * sizeof(double) , hipMemcpyHostToDevice )  ;
    hipMemcpy( dmat_in_device , dmat_in , mat.ncols * K * sizeof(double) , hipMemcpyHostToDevice ) ;
    //hipMemcpy( dmat_out_device, dmat_out, mat.nrows * K * sizeof(double) , hipMemcpyHostToDevice ) ;



    //Initialize the Grid and Block Dimension

    dim3 dimGrid( ceil(K / TILE_WIDTH) , ceil(mat.nrows/TILE_WIDTH) , 1  ) ;
    dim3 dimBlock(TILE_WIDTH , TILE_WIDTH , 1) ;

    hipEventRecord(startEvent, 0);

    dev_csr_spmm<<<dimGrid , dimBlock >>>(deviceCSRrow_indx, deviceCSRcol_id, deviceCSRvalues , dmat_in_device , dmat_out_device , K , mat.nrows) ;

    hipEventRecord(stopEvent, 0) ;
    hipEventSynchronize(stopEvent);

    float timeforKernel;
    hipEventElapsedTime(&timeforKernel, startEvent, stopEvent) ;

    printf("  Time for Kernel : %f\n",  timeforKernel);

    //hipDeviceSynchronize() ;
    //std::cout << "GPU out matrix before kernel\n";
    //print_dmat(dmat_out_GPU,  mat.nrows , K);

    //print_CSR(mat);

    hipMemcpy(dmat_out_GPU , dmat_out_device ,mat.nrows * K * sizeof(double) , hipMemcpyDeviceToHost ) ;


    hipEventRecord(stopEventMemKer, 0) ;

    hipEventSynchronize(startEventMemKer);
    hipEventSynchronize(stopEventMemKer);

    float timeforMemKernel;
    hipEventElapsedTime(&timeforMemKernel, startEventMemKer, stopEventMemKer) ;
    printf("  Time for Mem Cpy and Kernel : %f\n",  timeforMemKernel);

    //std::cout << "replace one argument to the below function with the values from gpu " << std::endl;
    //std::cout << "CPU\n";
    //print_dmat(dmat_out, mat.nrows , K);
    //std::cout << "GPU\n";
    print_dmat(dmat_out_GPU,  mat.nrows , K);
    check_dmat(dmat_out, dmat_out_GPU, mat.nrows, K);

    //Lets compute GFLOP
    unsigned int twoKnnz= 2 * K * mat.nnz ;
    printf("  2 * K * nnz : %d\n",  twoKnnz);


    float GFLOP = (twoKnnz / timeforMemKernel ) ;
    printf("  GFLOP : %d\n",  GFLOP);

    //print_dmat(dmat_out, mat.nrows, K);


    free(mat.row_indx);
    free(mat.col_id);
    free(mat.values);

    hipFree(deviceCSRrow_indx) ;
    hipFree(deviceCSRcol_id) ;
    hipFree(deviceCSRvalues) ;

    hipHostFree(pinnedMat.row_indx);
    hipHostFree(pinnedMat.col_id) ;
    hipHostFree(pinnedMat.values) ;

    //hipFree(device_nrows) ;
    //hipFree(device_ncols) ;
    //hipFree(device_nnz ) ;
    return 0;
}
