#include "hip/hip_runtime.h"
/*
 * spmm_csr_driver.cu
 * Copyright (C) 2018
 *  P Sadayappan (saday) <psaday@gmail.com>
 *  Aravind SUKUMARAN RAJAM (asr) <aravind_sr@outlook.com>
 *
 * Distributed under terms of the GNU LGPL3 license.
 */

#include "mm_helper.hpp"
#include "sparse_representation.hpp"
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define TILE_WIDTH 32

void check_dmat(double* a, double *b,  int n,  int K, bool quit_on_err = true ) {
    for ( int i = 0; i < n; ++i) {
        for ( int k = 0; k < K; ++k) {
            if(std::abs(a[i * K + k] - b[i * K + k]) > 1e-1) {
                std::cerr << "Possible error at " << i << std::endl;

                if(quit_on_err) {
                    exit(-1);
                }
            }
        }
    }

    if(quit_on_err)
        std::cout << "Verification succeeded\n";
    else
        std::cout << "Check error messages to see if verification succeeded. (No error msg == success)\n";
}

static  int g_seed = 0X4B1D;
inline int fastrand() {
    g_seed = (214013 * g_seed + 2531011);
    return (g_seed >> 16) & 0x7FFF;
}

void init_dmat(double *a,  int n,  int K, double offset) {
    for ( int i = 0; i < n; ++i) {
        for ( int k = 0; k < K; ++k) {
            a[i * K + k]  = i * K + k + offset;
            //a[i * K + j]  = fastrand() + offset;
        }
    }
}

void print_dmat(double *a,  int n,  int K) {
    for ( int i = 0; i < n; ++i) {
        for ( int j = 0; j < K; ++j) {
            std::cout << a[i * K + j]   << ' ';
        }
        std::cout << '\n';
    }
}

void print_CSR(CSR &mat) {
    for ( int r = 0; r < mat.nrows; ++r) {
         int row_start = mat.row_indx[r];
         int row_end = mat.row_indx[r + 1];
        for ( int j = row_start; j < row_end; ++j) {
             int col_id = mat.col_id[j];
            double val = mat.values[j];

	    std::cout << r << ' ' << col_id << ' ' <<  val << '\n';
        }
    }
}

void host_csr_spmm(CSR &mat, double * dmat_in, double * dmat_out,  int K) {
    for ( int r = 0; r < mat.nrows; ++r) {
         int row_start = mat.row_indx[r];
         int row_end = mat.row_indx[r + 1];

        for ( int k = 0; k < K; ++k) {
            dmat_out[r * K + k] = 0;
        }

        for ( int j = row_start; j < row_end; ++j) {
            int col_id = mat.col_id[j];
            double val = mat.values[j];

            for ( int k = 0; k < K; ++k) {
                dmat_out[r * K + k] += val * dmat_in[col_id * K + k];
            }
        }

    }
}

//Emin Code start
__global__ void dev_csr_spmm(unsigned int * deviceCSRrow_indx , unsigned int * deviceCSRcol_id  ,  double * deviceCSRvalues,
   double * dmat_in_device, double* dmat_out_device ,  int K , unsigned int device_nrows ){


      //int row= blockIdx.y*blockDim.y + threadIdx.y ;
      const int row=blockIdx.y;
      const int col= blockIdx.x * blockDim.x + threadIdx.x ;


      unsigned int numberOfRowCSR = device_nrows ;
      unsigned int colId;
      //const int row = blockIdx.x * blockDim.x + threadIdx.x ;
      //printf(" Rows = %d thread %d , block %d \n", numberOfRowCSR,  col , row);

      if ( (row < numberOfRowCSR) && (col < K) ) {

            printf(" thread %d , block %d \n",  col , row);

            double sum=0.0;

            // int row_start = A.row_indx[iy] ;
             unsigned int row_start = deviceCSRrow_indx[row];
            // int row_end = A.row_indx[iy + 1] ;
             unsigned int row_end = deviceCSRrow_indx[row+1] ;


            for (unsigned int i = row_start; i < row_end; i++) {
                  /* code */
                  //colId= A.col_id[i] ;
                  colId = deviceCSRcol_id[i] ;
                  double value = deviceCSRvalues[i] ;
                  //sum += A.values[i] * dmat_in_device[colId * K + ix] ;
                  //printf(" value %d  thread %d , block %d \n", value,  col , row);

                  sum +=  value * dmat_in_device[colId * K + col] ;

                  //std::cout << 'sum =' <<sum ;
                  //printf(" sum =  %d ,thread %d , block %d", sum, col , row);
            }
            __synctreads();
            //dmat_out[ix][iy] = sum ;
            printf(" sum = %d thread %d , block %d \n", sum,  col , row);
            dmat_out_device[row * K + col] = sum ;
            printf("dvice matrix %d\n", dmat_out_device[row * K + col] );
      }

}




int main(int argc, char *argv[]) {
    if(argc < 3) {
        std::cerr << "usage ./exec inputfile K  " << std::endl;
        exit(-1);
    }

     int K = std::atoi(argv[2]);
    CSR mat = read_matrix_market_to_CSR(argv[1]);
    //print_CSR(mat);
    std::cout << mat.nrows << ' ' << mat.ncols << ' ' << mat.nnz << ' ' << K << '\n';

    double *dmat_in = (double*)malloc(mat.ncols * K  * sizeof(double));
    double *dmat_out = (double*)malloc(mat.nrows * K * sizeof(double));
    double *dmat_out_GPU = (double*)malloc(mat.nrows * K * sizeof(double));

    init_dmat(dmat_in, mat.ncols, K,  1.0);
    print_dmat(dmat_in, mat.ncols, K);

    host_csr_spmm(mat, dmat_in, dmat_out, K);


    //Prepeare for Kernel
    //CSR *temMat;
    //temMat->nrows = mat.nrows ;
    //temMat.->ncols = mat.ncols ;
    //temMat.->nnz = mat.nnz ;

    unsigned int* deviceCSRrow_indx;
    unsigned int* deviceCSRcol_id;
    double* deviceCSRvalues;

     //int device_nrows;
     //int device_ncols;
     //int device_nnz;

     int a=1;

    hipMalloc((void**) &deviceCSRrow_indx ,(mat.nrows +1) * sizeof(unsigned int)) ;
    hipMalloc((void**) &deviceCSRcol_id , mat.ncols * sizeof(unsigned int)) ;
    hipMalloc((void**) &deviceCSRvalues , mat.nnz * sizeof(double)) ;

    //hipMalloc((void**) &device_nrows,  a*sizeof(int));
    //hipMalloc((void**) &device_ncols,  a*sizeof(int));
    //hipMalloc((void**) &device_nnz,  a*sizeof(int));

    //hipMalloc((void**) &(temMat->values) , mat.nnz * sizeof(double)) ;
    //hipMalloc((void**) &(temMat->row_indx) , mat.nrows * sizeof( int)) ;
    //hipMalloc((void**) &(temMat->col_id) , mat.ncols * sizeof( int)) ;

    //hipMalloc((void**) &(temMat->nrows) , sizeof( int)) ;
    //hipMalloc((void**) &(temMat->ncols) , sizeof( int)) ;
    //hipMalloc((void**) &(temMat->nnz) , sizeof( int)) ;

    //Initialize device addresses since it can not be accessed directly
    //hipMemcpy(temMat->values , mat.values , mat.nnz * sizeof(double) , hipMemcpyHostToDevice) ;
    //hipMemcpy(temMat->row_indx , mat.row_indx , mat.nrows * sizeof( int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(temMat->col_id , mat.col_id , mat.ncols * sizeof( int) , hipMemcpyHostToDevice) ;

    hipMemcpy(deviceCSRrow_indx , mat.row_indx ,  (mat.nrows+1) * sizeof(unsigned int) , hipMemcpyHostToDevice) ;
    hipMemcpy(deviceCSRcol_id, mat.col_id , mat.ncols * sizeof(unsigned int) , hipMemcpyHostToDevice) ;
    hipMemcpy(deviceCSRvalues , mat.values , mat.nnz * sizeof(double) , hipMemcpyHostToDevice) ;

    //hipMemcpy(device_nrows , mat.nrows , a*sizeof(int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(device_ncols , mat.ncols , a*sizeof(int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(device_nnz   , mat.nnz   , sizeof(int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(temMat->nrows , mat.nrows , 1*sizeof( int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(temMat->ncols , mat.ncols , 1*sizeof( int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(temMat->nnz , mat.nnz , 1*sizeof( int) , hipMemcpyHostToDevice) ;

    //CSR A;
    //hipMemcpyToSymbol(HIP_SYMBOL( A ), temMat , sizeof(CSR)) ;

    double *dmat_in_device ;
    hipMalloc((void**) &dmat_in_device , mat.ncols * K * sizeof(double)) ;

    double *dmat_out_device ;
    hipMalloc((void**) &dmat_out_device, mat.nrows * K * sizeof(double)) ;

    //copy to device
    hipMemcpy( dmat_in_device , dmat_in , mat.ncols * K * sizeof(double) , hipMemcpyHostToDevice ) ;
    hipMemcpy( dmat_out_device, dmat_out, mat.nrows * K * sizeof(double) , hipMemcpyHostToDevice ) ;


    //Initialize the Grid and Block Dimension

    dim3 dimGrid((K-1) / TILE_WIDTH + 1 , (mat.nrows -1)/1+1 , 1  ) ;
    dim3 dimBlock(TILE_WIDTH , 1 , 1) ;

    dev_csr_spmm<<<dimGrid , dimBlock>>>(deviceCSRrow_indx, deviceCSRcol_id, deviceCSRvalues , dmat_in_device , dmat_out_device , K , mat.nrows) ;
    hipDeviceSynchronize() ;
    //std::cout << "GPU out matrix before kernel\n";
    //print_dmat(dmat_out_GPU,  mat.nrows , K);

    print_CSR(mat);

    hipMemcpy(dmat_out_GPU , dmat_out_device ,mat.nrows * K * sizeof(double) , hipMemcpyDeviceToHost ) ;


    //td::cout << "replace one argument to the below function with the values from gpu " << std::endl;
    std::cout << "CPU\n";
    print_dmat(dmat_out, mat.nrows , K);
    std::cout << "GPU\n";
    print_dmat(dmat_out_GPU,  mat.nrows , K);
    check_dmat(dmat_out, dmat_out_GPU, mat.nrows, K);

    //print_dmat(dmat_out, mat.nrows, K);


    free(mat.row_indx);
    free(mat.col_id);
    free(mat.values);

    hipFree(deviceCSRrow_indx) ;
    hipFree(deviceCSRcol_id) ;
    hipFree(deviceCSRvalues) ;

    //hipFree(device_nrows) ;
    //hipFree(device_ncols) ;
    //hipFree(device_nnz ) ;
    return 0;
}
