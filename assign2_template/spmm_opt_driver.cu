#include "hip/hip_runtime.h"
/*
 * spmm_opt_driver.cu
 * Copyright (C) 2018
 *  P Sadayappan (saday) <psaday@gmail.com>
 *  Aravind SUKUMARAN RAJAM (asr) <aravind_sr@outlook.com>
 *
 * Distributed under terms of the GNU LGPL3 license.
 */

#include "mm_helper.hpp"
#include "sparse_representation.hpp"
#include <iostream>
#include <omp.h>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 32
#define MAX_BLOCK 50000
#define CHUNK_SIZE 1000

void check_dmat(double* a, double *b, unsigned int n, unsigned int K, bool quit_on_err = true ) {
    for (unsigned int i = 0; i < n; ++i) {
        for (unsigned int k = 0; k < K; ++k) {
            if(std::abs(a[i * K + k] - b[i * K + k]) > 1e-1) {
                std::cerr << "Possible error at " << i << std::endl;

                if(quit_on_err) {
                    exit(-1);
                }
            }
        }
    }

    if(quit_on_err)
        std::cout << "Verification succeeded\n";
    else
        std::cout << "Check error messages to see if verification succeeded. (No error msg == success)\n";
}

static unsigned int g_seed = 0X4B1D;
inline int fastrand() {
    g_seed = (214013 * g_seed + 2531011);
    return (g_seed >> 16) & 0x7FFF;
}

void init_dmat(double *a, unsigned int n, unsigned int K, double offset) {
    for (unsigned int i = 0; i < n; ++i) {
        for (unsigned int k = 0; k < K; ++k) {
            a[i * K + k]  = i * K + k + offset;
            //a[i * K + j]  = fastrand() + offset;
        }
    }
}

void print_dmat(double *a, unsigned int n, unsigned int K) {
    for (unsigned int i = 0; i < n; ++i) {
        for (unsigned int j = 0; j < K; ++j) {
            std::cout << a[i * K + j]   << ' ';
        }
        std::cout << '\n';
    }
}

void host_csr_spmm(CSR &mat, double * dmat_in, double * dmat_out, unsigned int K) {
    for (unsigned int r = 0; r < mat.nrows; ++r) {
        unsigned int row_start = mat.row_indx[r];
        unsigned int row_end = mat.row_indx[r + 1];

        for (unsigned int k = 0; k < K; ++k) {
            dmat_out[r * K + k] = 0;
        }

        for (unsigned int j = row_start; j < row_end; ++j) {
            unsigned int col_id = mat.col_id[j];
            double val = mat.values[j];

            for (unsigned int k = 0; k < K; ++k) {
                dmat_out[r * K + k] += val * dmat_in[col_id * K + k];
            }
        }

    }
}

//Emin Code start
__global__ void dev_opt_spmm_2(unsigned int * deviceCSRrow_indx , unsigned int * deviceCSRcol_id  ,  double * deviceCSRvalues,
   double * dmat_in_device, double* dmat_out_device ,  int K , unsigned int device_nrows ){

     __shared__ double vals[TILE_WIDTH] ;

      //int row= blockIdx.y*blockDim.y + threadIdx.y ;
      //const int thread_id_x=blockIdx.x * blockDim.x + threadIdx.x;
      //const int thread_id_y=blockIdx.y * blockDim.y + threadIdx.y;

      const int thread_id_x=blockIdx.x  * blockDim.x + threadIdx.x;

      //const int col= blockIdx.x * blockDim.x + threadIdx.x ;
      const int warp_id = thread_id_x /32 ;

      //const int irow= warp_id / K ;
      //onst int icol= warp_id & (K-1) ;

      int irow=warp_id ;
      int lane = thread_id_x & (31) ;


      unsigned int numberOfRowCSR = device_nrows ;

      //const int row = blockIdx.x * blockDim.x + threadIdx.x ;
      //printf(" Rows = %d thread %d , block %d \n", numberOfRowCSR,  col , row);

      if ( irow < numberOfRowCSR ) {

          for(int icol =0 ; icol < K ; icol++)
          {
                //printf(" icol %d , irow %d \n",  icol , irow);

                int colId;
                double sum=0;

                // int row_start = A.row_indx[iy] ;
                 unsigned int row_start = deviceCSRrow_indx[irow];

                 unsigned int row_end = deviceCSRrow_indx[irow+1] ;
                 //printf(" row_end = %d thread %d , block %d \n", row_end,  col , row);

                 //dmat_out_device[row * K + col] =0;
                 __syncthreads();
                 vals[threadIdx.x] = 0 ;
                    __syncthreads();

                 for ( int element = row_start + lane ; element < row_end; element+=32) {
                      /* code */

                      //colId= A.col_id[i] ;
                      colId = deviceCSRcol_id[element] ;
                      //printf(" colId = %d thread %d , block %d \n", colId,  col , row);

                      double value = deviceCSRvalues[element] ;
                      double value2 = dmat_in_device[colId * K + icol] ;

                      //printf(" colId = %d thread %d , block %d \n", colId,  threadIdx.x , irow);

                      //vals[threadIdx.x] += value + value2 ;
                        sum=value * value2;
                        atomicAdd(&vals[threadIdx.x] ,value * value2 );

                      //printf(" sum =  %d ,thread %d , block %d", sum, col , row);
                 }
                //Parallel Reduction
                __syncthreads();
                if(lane < 16)
                     atomicAdd(&vals[threadIdx.x] , vals[threadIdx.x + 16]) ;
                 if(lane < 8 )
                     atomicAdd(&vals[threadIdx.x] , vals[threadIdx.x + 8]) ;
                 if(lane < 4 )
                     atomicAdd(&vals[threadIdx.x] , vals[threadIdx.x + 4]) ;
                 if(lane < 2 )
                     atomicAdd(&vals[threadIdx.x] , vals[threadIdx.x + 2]) ;
                 if(lane < 1 )
                     atomicAdd(&vals[threadIdx.x] , vals[threadIdx.x + 1]) ;


                //__syncthreads();
                //dmat_out[ix][iy] = sum ;
                //printf(" sum = %d thread %d , block %d \n", sum,  col , row);
                __syncthreads();
                if(lane == 0)
                  atomicAdd(&dmat_out_device[irow * K + icol] , vals[threadIdx.x]) ;
                //printf("dvice matrix %d\n", dmat_out_device[row * K + col] );
          }
      }

}

//Emin Code start
__global__ void dev_opt_spmm(unsigned int * deviceCSRrow_indx , unsigned int * deviceCSRcol_id  ,  double * deviceCSRvalues,
   double * dmat_in_device, double* dmat_out_device ,  int K , unsigned int device_nrows ){

     __shared__ double vals[TILE_WIDTH] ;

      //int row= blockIdx.y*blockDim.y + threadIdx.y ;
      const int thread_id_x=blockIdx.x * blockDim.x + threadIdx.x;

      const int warp_id = thread_id_x /32 ;

      const int irow= warp_id / K ;
      const int icol= warp_id & (K-1) ;



      int lane = thread_id_x & (31) ;


      unsigned int numberOfRowCSR = device_nrows ;

      if ( irow < numberOfRowCSR && icol < K) {


            int colId;

            // int row_start = A.row_indx[iy] ;
             unsigned int row_start = deviceCSRrow_indx[irow];
             //printf(" row_start = %d thread %d , block %d \n", row_start,  col , row);
            // int row_end = A.row_indx[iy + 1] ;
             unsigned int row_end = deviceCSRrow_indx[irow+1] ;

             vals[threadIdx.x] = 0 ;

             for ( unsigned int element = row_start + lane ; element < row_end; element+=32) {
                  /* code */

                  //colId= A.col_id[i] ;
                  colId = deviceCSRcol_id[element] ;
                  //printf(" colId = %d thread %d , block %d \n", colId,  col , row);

                  double value = deviceCSRvalues[element] ;
                  double value2 = dmat_in_device[colId * K + icol] ;


                  atomicAdd(&vals[threadIdx.x] ,value * value2 );
                  //printf(" sum =  %d ,thread %d , block %d", sum, col , row);
             }
            //Parallel Reduction

            if(lane < 16)
                  atomicAdd(&vals[threadIdx.x] , vals[threadIdx.x + 16]) ;
            if(lane < 8 )
                  atomicAdd(&vals[threadIdx.x] , vals[threadIdx.x + 8]) ;
            if(lane < 4 )
                  atomicAdd(&vals[threadIdx.x] , vals[threadIdx.x + 4]) ;
            if(lane < 2 )
                  atomicAdd(&vals[threadIdx.x] , vals[threadIdx.x + 2]) ;
            if(lane < 1 )
                  atomicAdd(&vals[threadIdx.x] , vals[threadIdx.x + 1]) ;


            if(lane == 0)
                  atomicAdd(&dmat_out_device[irow * K + icol] , vals[threadIdx.x]) ;
            //printf("dvice matrix %d\n", dmat_out_device[row * K + col] );
      }

}


int main(int argc, char *argv[]) {
    if(argc < 3) {
        std::cerr << "usage ./exec inputfile K  " << std::endl;
        exit(-1);
    }

    unsigned int K = std::atoi(argv[2]);
    CSR mat = read_matrix_market_to_CSR(argv[1]);
    std::cout << mat.nrows << ' ' << mat.ncols << ' ' << mat.nnz << ' ' << K << '\n';

    double timeKernelCPUstart;
    double timeKernelCPUfinish;


    double *dmat_in = (double*)malloc(mat.ncols * K  * sizeof(double));
    double *dmat_out = (double*)malloc(mat.nrows * K * sizeof(double));

    init_dmat(dmat_in, mat.ncols, K,  1.0);

    /// No need to optimize host;
    host_csr_spmm(mat, dmat_in, dmat_out, K);

    //Cuda Events
    // events for timing
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent) ;
    hipEventCreate(&stopEvent)  ;


    hipEvent_t startEventMemKer , stopEventMemKer ;
    hipEventCreate(&startEventMemKer);
    hipEventCreate(&stopEventMemKer) ;

    //Lets implement pinned memory
    CSR pinnedMat;
    hipHostAlloc(&pinnedMat.row_indx , (mat.nrows +1)* sizeof(unsigned int), hipHostMallocMapped ) ;
    hipHostAlloc(&pinnedMat.col_id , mat.nnz * sizeof(unsigned int) , hipHostMallocMapped) ;
    hipHostAlloc(&pinnedMat.values , mat.nnz * sizeof(double), hipHostMallocMapped) ;

    memcpy(pinnedMat.row_indx , mat.row_indx ,(mat.nrows +1)* sizeof(unsigned int)) ;
    memcpy(pinnedMat.col_id , mat.col_id ,mat.nnz * sizeof(unsigned int) ) ;
    memcpy(pinnedMat.values , mat.values ,mat.nnz * sizeof(double)) ;

    pinnedMat.nrows=mat.nrows ;
    pinnedMat.ncols=mat.ncols ;
    pinnedMat.nnz = mat.nnz ;

    std::cout << mat.nrows << ' ' << mat.ncols << ' ' << mat.nnz << ' ' << K << '\n';

    //double *dmat_in = (double*)malloc(mat.ncols * K  * sizeof(double));
    //double *dmat_out = (double*)malloc(mat.nrows * K * sizeof(double));
    double *dmat_out_GPU = (double*)malloc(mat.nrows * K * sizeof(double));


    unsigned int* deviceCSRrow_indx;
    unsigned int* deviceCSRcol_id;
    double* deviceCSRvalues;


    hipMalloc((void**) &deviceCSRrow_indx ,(mat.nrows +1) * sizeof(unsigned int)) ;
    hipMalloc((void**) &deviceCSRcol_id , mat.nnz * sizeof(unsigned int)) ;
    hipMalloc((void**) &deviceCSRvalues , mat.nnz * sizeof(double)) ;

    double *dmat_in_device ;
    hipMalloc((void**) &dmat_in_device , mat.ncols * K * sizeof(double)) ;

    double *dmat_out_device ;
    hipMalloc((void**) &dmat_out_device, mat.nrows * K * sizeof(double)) ;

    hipEventRecord(startEventMemKer, 0);

    //hipMemcpy(deviceCSRrow_indx , mat.row_indx ,(mat.nrows+1) * sizeof(unsigned int) , hipMemcpyHostToDevice) ;
    //hipMemcpy(deviceCSRcol_id , mat.col_id , mat.nnz * sizeof(unsigned int) , hipMemcpyHostToDevice ) ;
    //hipMemcpy(deviceCSRvalues , mat.values , mat.nnz * sizeof(double) , hipMemcpyHostToDevice)  ;

    hipMemcpy(deviceCSRrow_indx , pinnedMat.row_indx ,(mat.nrows+1) * sizeof(unsigned int) , hipMemcpyHostToDevice );


    hipMemcpyAsync(deviceCSRcol_id , pinnedMat.col_id , mat.nnz * sizeof(unsigned int) , hipMemcpyHostToDevice ,0);
    hipMemcpyAsync(deviceCSRvalues , pinnedMat.values , mat.nnz * sizeof(double) , hipMemcpyHostToDevice ,0)  ;

    //copy to device
    hipMemcpyAsync( dmat_in_device , dmat_in , mat.ncols * K * sizeof(double) , hipMemcpyHostToDevice ,0) ;
    //hipMemcpy( dmat_out_device, dmat_out, mat.nrows * K * sizeof(double) , hipMemcpyHostToDevice ) ;

    int count = (mat.nrows- 1) / CHUNK_SIZE + 1;
    hipStream_t * stream = new hipStream_t[count] ;

    float min=10000000000000;
    float max=0;
    float average=0;
    float time;

    timeKernelCPUstart=omp_get_wtime( );

    for (int i = 0; i < count; i++) {
      /* code */

          hipStreamCreate(&stream[i]) ;

          hipEvent_t startTime, stopTime ;
          hipEventCreate(&startTime) ;
          hipEventCreate(&stopTime ) ;

          const int start = i * CHUNK_SIZE ;
          //const int end  = min(mat.nrows , (i +1) * CHUNK_SIZE) ;
          int end;
          if(mat.nrows < (i +1) * CHUNK_SIZE)
              end = mat.nrows;
          else
              end = (i +1) * CHUNK_SIZE ;

          hipEventRecord(startTime, 0);
          hipEventSynchronize(startTime);

          hipMemcpyAsync(deviceCSRrow_indx + start , pinnedMat.row_indx + start, (end - start +1 )* sizeof(unsigned int) , hipMemcpyHostToDevice, stream[i]) ;

          dim3 dimGrid( ( end -start  ) *K +1 , 1 ,  1  ) ; //for dev_opt_spmm

          //dim3 dimGrid(  end -start   +1 , 1 ,  1  ) ;
          dim3 dimBlock(TILE_WIDTH, 1 , 1) ; //

          dev_opt_spmm<<<dimGrid ,  dimBlock , 0, stream[i] >>>(deviceCSRrow_indx + start, deviceCSRcol_id, deviceCSRvalues , dmat_in_device , (dmat_out_device + start * K ), K , end-start); //

          hipMemcpyAsync( (dmat_out_GPU + start*K ), (dmat_out_device +start*K ), (end -start  ) * K * sizeof(double) , hipMemcpyDeviceToHost, stream[i] ) ;

          hipEventRecord(stopTime , 0) ;
          hipEventSynchronize(stopTime);
          hipEventElapsedTime(&time, startTime, stopTime) ;

          if(time > max)
              max=time;

          if(time < min)
              min=time;

          average = average + time;
    }

    for (int i = 0; i < count; i++) {
      hipStreamSynchronize(stream[i]) ;
      hipStreamDestroy(stream[i]);
    }

    timeKernelCPUfinish=omp_get_wtime( );
    check_dmat(dmat_out, dmat_out_GPU, mat.nrows, K);

    printf("  Time for Mem Cpy and Kernel : %f\n",  timeKernelCPUfinish);

    float timeforKernel;
    hipEventElapsedTime(&timeforKernel, startEvent, stopEvent) ;
    printf("  Time for Kernel : %f\n",  timeforKernel);



    //Lets compute GFLOP
    unsigned int twoKnnz= 2 * K * mat.nnz ;
    printf("  2 * K * nnz : %d\n",  twoKnnz);


    float GFLOP = (twoKnnz / (timeKernelCPUfinish-timeKernelCPUstart) )/1000000000 ;
    printf("  GFLOP : %f\n",  GFLOP);


    average=average/count;

    printf("Min Cuda Stream Event : %f\n",  min);
    printf("Max Cuda Stream Event : %f\n",  max);
    printf("Average Cuda Stream Event : %f\n",  average);



    free(mat.row_indx);
    free(mat.col_id);
    free(mat.values);

    hipFree(deviceCSRrow_indx) ;
    hipFree(deviceCSRcol_id) ;
    hipFree(deviceCSRvalues) ;

    hipHostFree(pinnedMat.row_indx);
    hipHostFree(pinnedMat.col_id) ;
    hipHostFree(pinnedMat.values) ;




    return 0;
}
